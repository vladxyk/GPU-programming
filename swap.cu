#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <sys/time.h>
#include <hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/swap.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>

#define CUDA_CHECK_RETURN(value) {\
hipError_t _m_cudaStat = value;\
if (_m_cudaStat != hipSuccess) {\
  fprintf(stderr, "Error %s at line %d in file %s\n",\
          hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
  exit(1);\
}\
}

using namespace std;
//a[1 2 3 4]
//b[5 6 7 8]
//swap
//a[5 6 7 8]
//b[1 2 3 4]

__global__ void swap(float *a, float *b, int vector_size){
    
	int indx = blockIdx.x * blockDim.x + threadIdx.x;

  float k = a[indx];
  a[indx] = b[indx];
  b[indx] = k;
}

float thrust_swap(int vector_size){

    thrust::host_vector<float> hA(vector_size);
    thrust::host_vector<float> hB(vector_size);

    for(int i = 0; i < vector_size; i++)
    {
        hA[i] = rand()%5;
        hB[i] = rand()%5;
    }

    thrust::device_vector<float> dA = hA;
    thrust::device_vector<float> dB = hB;

    cout << "THRUST_SWAP" << endl;
    /*
    cout << "vector A before swap : ";
    for (int i = 0; i < vector_size; i++){
        cout << hA[i] << " ";
    }

    cout << endl;

    cout << "vector B before swap : ";
    for (int i = 0; i < vector_size; i++){
      cout << hB[i] << " ";
    }
    cout << endl;
    */
    float ThrustTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    thrust::swap(dA, dB);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&ThrustTime, start, stop);

    thrust::copy(dA.begin(), dA.end(), hA.begin());
    thrust::copy(dB.begin(), dB.end(), hB.begin());

    /*
    cout << "vector A after swap: ";
    for (int i = 0; i < vector_size; i++){
        cout << hA[i] << " ";
    }

    cout << endl;

    cout << "vector B after swap: ";
    for (int i = 0; i < vector_size; i++){
      cout << hB[i] << " ";
    }
    cout << endl;
    */
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return ThrustTime;

}
float blas_swap(int vector_size){

    float *hA = new float[vector_size];
    float *hB = new float[vector_size];

    for(int i = 0; i < vector_size; i++)
    {
        hA[i] = rand()%5;
        hB[i] = rand()%5;
    }

    float *dA, *dB;
    
    CUDA_CHECK_RETURN(hipMalloc(&dA, sizeof(float) * vector_size));
    CUDA_CHECK_RETURN(hipMalloc(&dB, sizeof(float) * vector_size));

    CUDA_CHECK_RETURN(hipMemcpy(dA, hA, sizeof(float) * vector_size, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dB, hB, sizeof(float) * vector_size, hipMemcpyHostToDevice));

    cout << "CUBLAS_SWAP" << endl;
    /*
    cout << "vector A before swap: ";
    for (int i = 0; i < vector_size; i++){
        cout << hA[i] << " ";
    }

    cout << endl;

    cout << "vector B before swap : ";
    for (int i = 0; i < vector_size; i++){
      cout << hB[i] << " ";
    }
    cout << endl;
    */
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float CublasTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipblasSswap(handle, vector_size, dA, 1, dB, 1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

    hipEventElapsedTime(&CublasTime, start, stop);

    CUDA_CHECK_RETURN(hipMemcpy(hA, dA, sizeof(float) * vector_size, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(hB, dB, sizeof(float) * vector_size, hipMemcpyDeviceToHost));
    
    /*
    cout << "vector A after swap : ";
    for (int i = 0; i < vector_size; i++){
        cout << hA[i] << " ";
    }

    cout << endl;

    cout << "vector B after swap : ";
    for (int i = 0; i < vector_size; i++){
      cout << hB[i] << " ";
    }
    cout << endl;
    */
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(handle);
    delete [] hA;
    delete [] hB;
    hipFree(dA);
    hipFree(dB);

    return CublasTime;
}

int main(int argc, char *argv[])
{
    cout << "1 arg - vector_size, 2 arg - block_size" << endl << endl;

    int vector_size = atoi(argv[1]);
    int block_size = atoi(argv[2]);
        
    srand(time(NULL));
    
    float *hA = new float[vector_size];
    float *hB = new float[vector_size];

    for(int i = 0; i < vector_size; i++)
    {
        hA[i] = rand()%5;
        hB[i] = rand()%5; 
    }

    float *dA, *dB;
    
    CUDA_CHECK_RETURN(hipMalloc(&dA, sizeof(float) * vector_size));
    CUDA_CHECK_RETURN(hipMalloc(&dB, sizeof(float) * vector_size));

    CUDA_CHECK_RETURN(hipMemcpy(dA, hA, sizeof(float) * vector_size, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dB, hB, sizeof(float) * vector_size, hipMemcpyHostToDevice));

    cout << "CUDA_SWAP" << endl;
   /*
    cout << "vector A before swap : ";
    for (int i = 0; i < vector_size; i++){
        cout << hA[i] << " ";
    }

    cout << endl;

    cout << "vector B before swap : ";
    for (int i = 0; i < vector_size; i++){
      cout << hB[i] << " ";
    }
    cout << endl;
    */
    int num_blocks = (int)ceil((float)vector_size / block_size);
    
    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    swap <<<num_blocks, block_size>>> (dA, dB, vector_size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

    hipEventElapsedTime(&elapsedTime, start, stop);

    CUDA_CHECK_RETURN(hipMemcpy(hA, dA, sizeof(float) * vector_size, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(hB, dB, sizeof(float) * vector_size, hipMemcpyDeviceToHost));
  
    //cout << "CUDA_COPY" << endl;
  /* 
    cout << "vector A after swap : ";
    for (int i = 0; i < vector_size; i++){
        cout << hA[i] << " ";
    }

    cout << endl;

    cout << "vector B after swap : ";
    for (int i = 0; i < vector_size; i++){
      cout << hB[i] << " ";
    }
    cout << endl;
   */
    cout << "Cuda_Time = " << elapsedTime << endl;
    cout << endl;
    
    float cublas = blas_swap(vector_size);
    cout << "Cublas_Time = " << cublas << endl;
    cout << endl;
  

    float thrust = thrust_swap(vector_size);
    cout << "Thrust_Time = " << thrust << endl;
    cout << endl;
  
    hipEventDestroy(start);
    hipEventDestroy(stop);
    delete [] hA;
    delete [] hB;
    hipFree(dA);
    hipFree(dB);
}
