#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <sys/time.h>
#include <hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>

#define CUDA_CHECK_RETURN(value) {\
hipError_t _m_cudaStat = value;\
if (_m_cudaStat != hipSuccess) {\
  fprintf(stderr, "Error %s at line %d in file %s\n",\
          hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
  exit(1);\
}\
}

using namespace std;


__global__ void Copy(float *a, float *b, int vector_size){
    
	int indx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (indx < vector_size){
    	b[indx] = a[indx]; 
    } 
}
float thrust_copy(int vector_size){

    thrust::host_vector<float> hA(vector_size);
    thrust::host_vector<float> hB(vector_size);

    for(int i = 0; i < vector_size; i++)
    {
        hA[i]=i;
    }

    thrust::device_vector<float> dA = hA;
    thrust::device_vector<float> dB = hB;

    float ThrustTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    thrust::copy(dA.begin(), dA.end(), dB.begin());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&ThrustTime, start, stop);

    thrust::copy(dB.begin(), dB.end(), hB.begin());

    cout << "THRUST_COPY" << endl;
/*    
    cout << "vector A : ";
    for (int i = 0; i < vector_size; i++){
        cout << hA[i] << " ";
    }

    cout << endl;

    cout << "vector B : ";
    for (int i = 0; i < vector_size; i++){
      cout << hB[i] << " ";
    }
    cout << endl;
*/    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return ThrustTime;

}
float blas_copy(int vector_size){

    float *hA = new float[vector_size];
    float *hB = new float[vector_size];

    for(int i = 0; i < vector_size; i++)
    {
        hA[i]=i;
    }

    float *dA, *dB;
    
    CUDA_CHECK_RETURN(hipMalloc(&dA, sizeof(float) * vector_size));
    CUDA_CHECK_RETURN(hipMalloc(&dB, sizeof(float) * vector_size));

    CUDA_CHECK_RETURN(hipMemcpy(dA, hA, sizeof(float) * vector_size, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dB, hB, sizeof(float) * vector_size, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float CublasTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipblasScopy(handle, vector_size, dA, 1, dB, 1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

    hipEventElapsedTime(&CublasTime, start, stop);

    CUDA_CHECK_RETURN(hipMemcpy(hB, dB, sizeof(float) * vector_size, hipMemcpyDeviceToHost));
    
    cout << "CUBLAS_COPY" << endl;
/*    
    cout << "vector A : ";
    for (int i = 0; i < vector_size; i++){
        cout << hA[i] << " ";
    }

    cout << endl;

    cout << "vector B : ";
    for (int i = 0; i < vector_size; i++){
      cout << hB[i] << " ";
    }
    cout << endl;
*/    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(handle);
    delete [] hA;
    delete [] hB;
    hipFree(dA);
    hipFree(dB);

    return CublasTime;
}

int main(int argc, char *argv[])
{
    cout << "1 arg - vector_size, 2 arg - block_size" << endl << endl;

    int vector_size = atoi(argv[1]);
    int block_size = atoi(argv[2]);
    
    srand(time(NULL));
    
    float *hA = new float[vector_size];
    float *hB = new float[vector_size];

    for(int i = 0; i < vector_size; i++)
    {
        hA[i]=i;
    }

    float *dA, *dB;
    
    CUDA_CHECK_RETURN(hipMalloc(&dA, sizeof(float) * vector_size));
    CUDA_CHECK_RETURN(hipMalloc(&dB, sizeof(float) * vector_size));

    CUDA_CHECK_RETURN(hipMemcpy(dA, hA, sizeof(float) * vector_size, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dB, hB, sizeof(float) * vector_size, hipMemcpyHostToDevice));

    int num_blocks = (int)ceil((float)vector_size / block_size);
    
    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    Copy <<<num_blocks, block_size>>> (dA, dB, vector_size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

    hipEventElapsedTime(&elapsedTime, start, stop);

    CUDA_CHECK_RETURN(hipMemcpy(hB, dB, sizeof(float) * vector_size, hipMemcpyDeviceToHost));
    
    cout << "CUDA_COPY" << endl;
/*    
    cout << "vector A : ";
    for (int i = 0; i < vector_size; i++){
        cout << hA[i] << " ";
    }

    cout << endl;

    cout << "vector B : ";
    for (int i = 0; i < vector_size; i++){
      cout << hB[i] << " ";
    }
    cout << endl;
*/    
    cout << "Cuda_Time = " << elapsedTime << endl;
    cout << endl;
    
    float cublas = blas_copy(vector_size);
    cout << "Cublas_Time = " << cublas << endl;
    cout << endl;

    float thrust = thrust_copy(vector_size);
    cout << "Thrust_Time = " << thrust << endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    delete [] hA;
    delete [] hB;
    hipFree(dA);
    hipFree(dB);
}
